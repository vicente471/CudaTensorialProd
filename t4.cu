/*******************************************************************************
 *
 * t4.cu: Producto tensorial de matrices con CUDA
 *
 * Programmer: Cristobal Gallardo & Vicente Santos
 *
 * Santiago de Chile, 7/12/2023
 *
 ******************************************************************************/



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h> 
#include <sys/types.h>
#include <unistd.h>
#include <time.h>

#define THREADxBLOCK 1024

__global__ void Process(float *a, float *b, float *res, int r, int c, int rb, int cb) {      //Calculo del producto tensorial
    int index = blockIdx.x * blockDim.x + threadIdx.x, i, j, m, n;

    if (index < r * rb * c* cb) {
        i = (index / (cb * rb)) / c;
        j = (index / (cb * rb)) % c;
        m = (index/ cb) % rb;
        n = index % cb;
        res[index] = a[(i * c) + j] * b[(m * cb) + n];
    }
}


float *ReadMatrix(unsigned int r, unsigned int c) {            //Lee la matriz
    unsigned int i, j;
    float *mat = (float *)malloc(r * c * sizeof(float));
    for (i = 0; i < r; i = i + 1) {
        for (j = 0; j < c; j = j + 1) {
            scanf("%f", &mat[i * c + j]);
        }
    }
    return mat;
}

void PrintMatrix(unsigned int r, unsigned int c, float *mat) {       //Imprime la matriz
    unsigned int i, j;
    for (i = 0; i < r; i = i + 1) {
        for (j = 0; j < c; j = j + 1) {
            printf(" %10.2f ", mat[i * c + j]);
        }
        printf("\n");
    }
}



void Usage(char *mess) {

    printf("\nUsage: %s -M -O < data.txt\n",mess);
    printf("M = {B: procesamiento solo con bloques, T: procesamiento con bloques y hebras}\n");
    printf("O = {S: modo silencioso, V: modo vervoso}\n\n");
}

int main(int argc, char **argv){
    int m, k, n, mkkn;
    float *Matrix1, *Matrix2, *MatrixC, *Matrix1_1D, *Matrix2_1D, *MatrixC_1D, E_cpu;
    long E_wall;
    time_t  ts, te;
    clock_t cs, ce;
    ts = time(NULL);
    cs = clock();
   
   
    if (argc == 3){
        scanf("%d",&m);
        scanf("%d",&k);
        scanf("%d",&n);
        mkkn = m * k * k * n;
        printf("m = %d k = %d n = %d\n", m, k, n);
        Matrix1 = ReadMatrix(m, k);
        Matrix2 = ReadMatrix(k, n); 
        if (strcmp(argv[2], "-V") == 0) {            //Se imprimen las matrices
                printf(" Matriz A(%d,%d):\n\n", m, k); 
                PrintMatrix(m, k, Matrix1);
                printf("\n");
                printf(" Matriz B(%d,%d):\n\n", k, n);
                PrintMatrix(k, n, Matrix2);
                
        }
        hipMalloc((void**)&Matrix1_1D, m * k * sizeof(float));             //Se asignan tamanos
        hipMalloc((void**)&Matrix2_1D, k * n * sizeof(float));
        hipMalloc((void**)&MatrixC_1D, m * k * k * n * sizeof(float));
        MatrixC = (float *)malloc(mkkn * sizeof(float *));
       
        hipMemcpy(Matrix1_1D, Matrix1, m * k * sizeof(float), hipMemcpyHostToDevice);     //Se envian al Device
        hipMemcpy(Matrix2_1D, Matrix2, k * n * sizeof(float), hipMemcpyHostToDevice);
        
        if (strcmp(argv[1], "-B") == 0){                          //Calculo solo bloques
            //Modo con solo Bloques
            Process<<<mkkn, 1>>>(Matrix1_1D, Matrix2_1D, MatrixC_1D, m, k, k, n);
        }
        
        if (strcmp(argv[1], "-T") == 0){                        //Calculo con hilos y bloques
            //Modo con Bloques y hebras
            Process<<<(mkkn + (THREADxBLOCK - 1)) / THREADxBLOCK, THREADxBLOCK>>>(Matrix1_1D, Matrix2_1D, MatrixC_1D, m, k, k, n);
        }
        
        hipMemcpy(MatrixC, MatrixC_1D, m * k * k * n *sizeof(float), hipMemcpyDeviceToHost);       //Se recibe el resultado desde el dispositivo
        
        if (strcmp(argv[2], "-V") == 0) {       //Se muestra el resultado en pantalla
            printf(" Matriz resultado(%d,%d):\n\n", m * k, k * n);
            PrintMatrix(m * k, k * n, MatrixC);
        }
        

        
    }else{
        Usage(argv[0]);
    }
    ce = clock();
    te = time(NULL);
    E_wall = (long) (te - ts);
    E_cpu = (float)(ce - cs) / CLOCKS_PER_SEC;
    printf(" Elapsed CPU Time %f Wall Time %ld \n", E_cpu, E_wall); 
    // Liberar memoria en el host
    free(Matrix1);
    free(Matrix2);
    free(MatrixC);

    // Libera memoria del Device
    hipFree(Matrix1_1D);
    hipFree(Matrix2_1D);
    hipFree(MatrixC_1D);
    
        
    return 0;
}